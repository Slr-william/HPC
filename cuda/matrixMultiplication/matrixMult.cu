#include <iostream>
#include <string>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <fstream>

using namespace std;

__global__ void MatrixMulKernel(float *d_M, float *d_N, float *d_P,int width){
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;
	
	if ((Row < width)&&(Col < width)){
		float Pvalue = 0;
		for (int i = 0; i < width; ++i){
			Pvalue += d_M[Row*width+i]*d_N[i*width+Col];
		}
		d_P[Row*width + Col] = Pvalue;
	}
}

int matrixMulHost(float *h_M, float *h_N, float *h_P, int width){
    int Pvalue;

    for(int row = 0; row < width ; ++row){
        for(int col = 0; col < width ; ++col){
            Pvalue = 0;
            for(int k = 0; k < width ; ++k){
                Pvalue += h_M[row*width+k] * h_N[k*width+col];
            }
            h_P[row*width+col] = Pvalue;
        }
    }
    return 0;
}

int initValues(float *data, int width){
    for(int i = 0; i < width*width; i++)
        data[i] = 2;
    return 0;
}

int main(int argc, char const *argv[])
{
	float *h_M, *h_N, *h_P,*h_P_d;
    float *d_M, *d_N,*d_P;
    std::string num = argv[1];
    int width = std::stoi(num);
    int size = width * width * sizeof(float);
    clock_t start, end, startGPU, endGPU;
    double cpu_time_used, gpu_time_used, aceleration;

    for (int times = 0; times < 20; times++){
        h_M = (float*)malloc(size);
        h_N = (float*)malloc(size);
        h_P = (float*)malloc(size);
        h_P_d = (float*)malloc(size);

        initValues(h_M, width);
        initValues(h_N, width);

        /////////Algoritmo Secuencial////////////////////////////////////////////
        start = clock();
        matrixMulHost(h_M, h_N, h_P, width);
        end = clock();
        cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
        printf("Tiempo algoritmo secuencial: %.10f\n", cpu_time_used);
        /////////Algoritmo Secuencial/////////////////////////////////////////////

        hipMalloc((void**)&d_M,size);
        hipMalloc((void**)&d_N,size);
        hipMalloc((void**)&d_P,size);
        //////////////////////Algoritmo Paralelo///////////////////////////
        startGPU = clock();
        hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);
        hipMemcpy(d_N, h_N, size, hipMemcpyHostToDevice);

        int blockSize = 32;
        dim3 dimBlock(blockSize,blockSize,1);
        dim3 dimGrid(ceil(width/float(blockSize)),ceil(width/float(blockSize)),1);
        MatrixMulKernel<<<dimGrid,dimBlock>>>(d_M,d_N,d_P,width);
        hipDeviceSynchronize();
        hipMemcpy(h_P_d,d_P,size,hipMemcpyDeviceToHost);
        endGPU = clock();
        gpu_time_used = ((double) (endGPU - startGPU)) / CLOCKS_PER_SEC;
        aceleration = cpu_time_used/gpu_time_used;
        printf("Tiempo algoritmo paralelo: %.10f\n", gpu_time_used);
        printf("La aceleración obtenida es de %.10fX\n",aceleration);

        std::string name =  "TimesMult.txt"+num;

        ofstream outfile(name,ios::binary | ios::app);
        outfile << gpu_time_used<<" "<< cpu_time_used <<" "<< aceleration << "\n";
        outfile.close();

        free(h_M);
        free(h_N);
        free(h_P);
        hipFree(d_M);
        hipFree(d_N);
        hipFree(d_P);
    }

	return 0;
}

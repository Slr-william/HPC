#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp> // import no include errors 
#include <fstream>
#include <string>

#define RED 2
#define GREEN 1
#define BLUE 0

using namespace cv;
using namespace std;

#define MASK_WIDTH 9

__constant__ float constMask[MASK_WIDTH];

__device__ unsigned char setNumber(int value){
    if(value < 0)
        value = 0;
    else
        if(value > 255)
            value = 255;
    return (unsigned char)value;
}


__global__ void sobelFilter(unsigned char *imageInput, int width, int height, unsigned int maskWidth, unsigned char *imageOutput){
    unsigned int row = blockIdx.y*blockDim.y+threadIdx.y;
    unsigned int col = blockIdx.x*blockDim.x+threadIdx.x;

    int Pvalue = 0;

    int N_start_point_row = row - (maskWidth/2);
    int N_start_point_col = col - (maskWidth/2);

    for(int i = 0; i < maskWidth; i++){
        for(int j = 0; j < maskWidth; j++ ){
            if((N_start_point_col + j >=0 && N_start_point_col + j < width) && (N_start_point_row + i >=0 && N_start_point_row + i < height)){
                Pvalue += imageInput[(N_start_point_row + i)*width+(N_start_point_col + j)] * constMask[i*maskWidth+j];
            }
        }
    }
    imageOutput[row*width+col] = setNumber(Pvalue);
}

__global__ void img2gray(unsigned char *imageInput, int width, int height, unsigned char *imageOutput){
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if((row < height) && (col < width)){
        imageOutput[row*width+col] = imageInput[(row*width+col)*3+RED]*0.299 + imageInput[(row*width+col)*3+GREEN]*0.587 
        + imageInput[(row*width+col)*3+BLUE]*0.114;
    }
}


int main(int argc, char **argv){
    hipError_t error = hipSuccess;
    clock_t start, end;
    int times = 1;
    double cpu_time_used;
    float h_Mask[] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
    unsigned char *h_dataImage, *d_dataImage, *d_imageOutput, *h_imageOutput, *d_sobelOutput;
    hipEvent_t startGPU, stopGPU;
    hipEventCreate(&startGPU);
    hipEventCreate(&stopGPU);
    int maskWidth = MASK_WIDTH;

    if(argc !=3 || argc != 4){
        printf("Enter the image's name and to repeat (op w)\n");
        return -1;
    }
    
    bool writeImage = false;
    if (argc == 4){
        writeImage = true;
    }

    char* imageName = argv[1];
    times = atoi(argv[2]);

    Mat image;
    image = imread(imageName, 1);

    if(!image.data){return -1;}

    Size s = image.size();

    int width = s.width;
    int height = s.height;
    int size = sizeof(unsigned char)*width*height*image.channels();
    int sizeGray = sizeof(unsigned char)*width*height;

    string text  = string(imageName)+"CMTimes";

    for (int i = 0; i < times; i++)
    {
        h_dataImage = (unsigned char*)malloc(size);
        error = hipMalloc((void**)&d_dataImage, size);
        if(error != hipSuccess){printf("Error-> memory allocation of d_dataImage\n");exit(-1);}

        h_imageOutput = (unsigned char *)malloc(sizeGray);
        error = hipMalloc((void**)&d_imageOutput, sizeGray);
        if(error != hipSuccess){printf("Error-> memory allocation of d_imageOutput\n");exit(-1);}

        error = hipMalloc((void**)&d_sobelOutput, sizeGray);
        if(error != hipSuccess){printf("Error-> memory allocation of d_sobelOutput\n");exit(-1);}

        h_dataImage = image.data;

        error = hipMemcpy(d_dataImage, h_dataImage, size, hipMemcpyHostToDevice);
        if(error != hipSuccess){printf("Error sending data from host to device in dataImage\n");exit(-1);}

        error = hipMemcpyToSymbol(HIP_SYMBOL(constMask), h_Mask, maskWidth*sizeof(float));
	if(error != hipSuccess){printf("Error in const memory\n");exit(-1);}

        int blockSize = 32;
        dim3 dimBlock(blockSize, blockSize, 1);
        dim3 dimGrid(ceil(width/float(blockSize)), ceil(height/float(blockSize)), 1);
        img2gray<<<dimGrid, dimBlock>>>(d_dataImage, width, height, d_imageOutput);
        hipDeviceSynchronize();

        hipEventRecord(startGPU);
        sobelFilter<<<dimGrid, dimBlock>>>(d_imageOutput, width, height, maskWidth, d_sobelOutput);
        hipDeviceSynchronize();
        hipEventRecord(stopGPU);

        error = hipMemcpy(h_imageOutput, d_sobelOutput, sizeGray, hipMemcpyDeviceToHost);
        if(error != hipSuccess){printf("Error sending data from device to host in imageOutput\n");exit(-1);}
        hipEventSynchronize(stopGPU);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, startGPU, stopGPU);

        Mat gray_image;
        gray_image.create(height, width, CV_8UC1);
        gray_image.data = h_imageOutput;

        start = clock();
        Mat gray_image_opencv, grad_x, abs_grad_x;
        cvtColor(image, gray_image_opencv, CV_BGR2GRAY);
        Sobel(gray_image_opencv, grad_x, CV_8UC1, 1, 0, 3, 1, 0, BORDER_DEFAULT);
        convertScaleAbs(grad_x, abs_grad_x);
        end = clock();


        if (writeImage){
            imwrite("./SMsobel.jpg", gray_image);
            writeImage = false;
        }

        //namedWindow(imageName, WINDOW_NORMAL);
        //namedWindow("Gray Image CUDA", WINDOW_NORMAL);
        //namedWindow("Sobel Image OpenCV", WINDOW_NORMAL);

        //imshow(imageName, image);
        //imshow("Gray Image CUDA", gray_image);
        //imshow("Sobel Image OpenCV", abs_grad_x);

        //waitKey(0);

        //free(h_dataImage);
        //free(h_imageOutput);
        cpu_time_used = ((double) (end - start)) /CLOCKS_PER_SEC;
        printf("Time in CPU: %.10f, time in GPU: %.10f\n", cpu_time_used, milliseconds);

        ofstream outfile(text.c_str(),ios::binary | ios::app);
        outfile << cpu_time_used*1000 <<", "<< milliseconds << "\n";
        outfile.close();

        hipFree(d_dataImage);
        hipFree(d_imageOutput);
        hipFree(d_sobelOutput);
    }
    return 0;
}

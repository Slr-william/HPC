#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp> // import no include errors 
#include <opencv2/gpu/gpu.hpp>
#include <fstream>
#include <string>

#define RED 2
#define GREEN 1
#define BLUE 0

using namespace cv;
using namespace std;

int main(int argc, char const *argv[])
{
	if(argc !=3 || argc != 4){
        printf("Enter the image's name and to repeat \n");
        return -1;
    }
    hipError_t error = hipSuccess;
    clock_t start, end;
    int times = 1;
    bool writeImage = false;
    hipEvent_t startGPU, stopGPU;
    hipEventCreate(&startGPU);
    hipEventCreate(&stopGPU);

    if (argc == 4){
        writeImage = true;
    }

    char* imageName = argv[1];
    times = atoi(argv[2]);

	Mat src = imread("car1080.jpg", 0);
	if (!src.data) exit(1);
	gpu::GpuMat d_src(src);
	gpu::GpuMat d_dst;
	gpu::Sobel(d_src, d_dst, CV_8UC1, 1, 0, 3, 1, 0, BORDER_DEFAULT);
	Mat dst(d_dst);
	imwrite("opencvCudaSobel.jpg", dst);
	return 0;
}